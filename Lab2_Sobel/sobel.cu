#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE  8

unsigned char *image_s = NULL;     // source image array
unsigned char *image_t = NULL;     // target image array
FILE *fp_s = NULL;                 // source file handler
FILE *fp_t = NULL;                 // target file handler

unsigned int   width, height;      // image width, image height
unsigned int   rgb_raw_data_offset;// RGB raw data offset
unsigned char  bit_per_pixel;      // bit per pixel
unsigned short byte_per_pixel;     // byte per pixel

// bitmap header
unsigned char header[54] = {
	0x42,        // identity : B
	0x4d,        // identity : M
	0, 0, 0, 0,  // file size
	0, 0,        // reserved1
	0, 0,        // reserved2
	54, 0, 0, 0, // RGB data offset
	40, 0, 0, 0, // struct BITMAPINFOHEADER size
	0, 0, 0, 0,  // bmp width
	0, 0, 0, 0,  // bmp height
	1, 0,        // planes
	24, 0,       // bit per pixel
	0, 0, 0, 0,  // compression
	0, 0, 0, 0,  // data size
	0, 0, 0, 0,  // h resolution
	0, 0, 0, 0,  // v resolution 
	0, 0, 0, 0,  // used colors
	0, 0, 0, 0   // important colors
};

// sobel mask (5x5 version)
// Task 2: Put mask[][][] into Shared Memroy
int
mask[MASK_N][MASK_X][MASK_Y] = {
	{{ -1, -4, -6, -4, -1},
	 { -2, -8,-12, -8, -2},
	 {  0,  0,  0,  0,  0},
	 {  2,  8, 12,  8,  2},
	 {  1,  4,  6,  4,  1}}
,
	{{ -1, -2,  0,  2,  1},
	 { -4, -8,  0,  8,  4},
	 { -6,-12,  0, 12,  6},
	 { -4, -8,  0,  8,  4},
	 { -1, -2,  0,  2,  1}}
};

int
read_bmp (const char *fname_s) {
	fp_s = fopen(fname_s, "rb");
	if (fp_s == NULL) {
		printf("fopen fp_s error\n");
		return -1;
	}

	// move offset to 10 to find rgb raw data offset
	fseek(fp_s, 10, SEEK_SET);
	fread(&rgb_raw_data_offset, sizeof(unsigned int), 1, fp_s);

	// move offset to 18 to get width & height;
	fseek(fp_s, 18, SEEK_SET); 
	fread(&width,  sizeof(unsigned int), 1, fp_s);
	fread(&height, sizeof(unsigned int), 1, fp_s);

	// get bit per pixel
	fseek(fp_s, 28, SEEK_SET); 
	fread(&bit_per_pixel, sizeof(unsigned short), 1, fp_s);
	byte_per_pixel = bit_per_pixel / 8;

	// move offset to rgb_raw_data_offset to get RGB raw data
	fseek(fp_s, rgb_raw_data_offset, SEEK_SET);

	// Task 3: Assign image_s to Pinnned Memory //Done!
	// Hint  : err = cudaMallocHost ( ... )
	//         if (err != CUDA_SUCCESS)
	int err;
	err = hipHostMalloc(&image_s,(size_t)width * height * byte_per_pixel, hipHostMallocDefault);
	if (err != hipSuccess) {
		printf("malloc images_s error\n");
		return -1;
	}
	// Task 3: Assign image_t to Pinned Memory //Done!
	// Hint  : err = cudaMallocHost ( ... )
	//         if (err != CUDA_SUCCESS)
	err = hipHostMalloc(&image_t, (size_t) width * height * byte_per_pixel, hipHostMallocDefault);
	if (err != hipSuccess) {
		printf("malloc image_t error\n");
		return -1;
	}

	fread(image_s, sizeof(unsigned char), (size_t)(long) width * height * byte_per_pixel, fp_s);

	return 0;
}

__global__ void sobel(unsigned char *image_s, unsigned char *image_t, 
                        unsigned int width, unsigned int height,
                        unsigned short byte_per_pixel,
                        int* d_mask ) {
	int  x, y, i, v, u;            // for loop counter
	int  R, G, B;                  // color of R, G, B
	double val[MASK_N*3] = {0.0};
	int adjustX, adjustY, xBound, yBound;

    // Task 2: Put mask[][][] into Shared Memory
	// Hint  : Please declare it in kernel function
	//         Then use some threads to move data from global memory to shared memory
	//         Remember to __syncthreads() after it's done <WHY?>
	__shared__ int mask[MASK_N][MASK_X][MASK_Y];
	v = threadIdx.x;
	if(v < MASK_X){
		for(int i = 0; i < MASK_N; i++){
			for(int j = 0; j < MASK_Y; j++){
				mask[i][v][j] = d_mask[i * MASK_X * MASK_Y + v * MASK_Y + j];
			}
		}
	}		
	__syncthreads();
	// Task 1: Relabel x, y into combination of blockIdx, threadIdx ... etc
	// Hint A: We do not have enough threads for each pixels in the image, so what should we do?
	// Hint B: Maybe you can map each y to different threads in different blocks
	for (y = blockIdx.x; y < blockIdx.x + 1; ++y) {
		for (x = threadIdx.x; x < width; x+=256) {
			for (i = 0; i < MASK_N; ++i) {
				adjustX = (MASK_X % 2) ? 1 : 0;
				adjustY = (MASK_Y % 2) ? 1 : 0;
				xBound = MASK_X /2;
				yBound = MASK_Y /2;

				val[i*3+2] = 0.0;
				val[i*3+1] = 0.0;
				val[i*3] = 0.0;

				for (v = -yBound; v < yBound + adjustY; ++v) {
					for (u = -xBound; u < xBound + adjustX; ++u) {
						if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
							R = image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 2];
							G = image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 1];
							B = image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 0];
							val[i*3+2] += R * mask[i][u + xBound][v + yBound];
							val[i*3+1] += G * mask[i][u + xBound][v + yBound];
							val[i*3+0] += B * mask[i][u + xBound][v + yBound];
						}	
					}
				}
			}

			double totalR = 0.0;
			double totalG = 0.0;
			double totalB = 0.0;
			for (i = 0; i < MASK_N; ++i) {
				totalR += val[i*3+2] * val[i*3+2];
				totalG += val[i*3+1] * val[i*3+1];
				totalB += val[i*3+0] * val[i*3+0];
			}

			totalR = sqrt(totalR) / SCALE;
			totalG = sqrt(totalG) / SCALE;
			totalB = sqrt(totalB) / SCALE;
			const unsigned char cR = (totalR > 255.0) ? 255 : totalR;
			const unsigned char cG = (totalG > 255.0) ? 255 : totalG;
			const unsigned char cB = (totalB > 255.0) ? 255 : totalB;
			image_t[ byte_per_pixel * (width * y + x) + 2 ] = cR;
			image_t[ byte_per_pixel * (width * y + x) + 1 ] = cG;
			image_t[ byte_per_pixel * (width * y + x) + 0 ] = cB;
		}
	}

	return;
}

int
write_bmp (const char *fname_t) {
	unsigned int file_size; // file size

	fp_t = fopen(fname_t, "wb");
	if (fp_t == NULL) {
		printf("fopen fname_t error\n");
		return -1;
	}

	// file size  
	file_size = width * height * byte_per_pixel + rgb_raw_data_offset;
	header[2] = (unsigned char)(file_size & 0x000000ff);
	header[3] = (file_size >> 8)  & 0x000000ff;
	header[4] = (file_size >> 16) & 0x000000ff;
	header[5] = (file_size >> 24) & 0x000000ff;

	// width
	header[18] = width & 0x000000ff;
	header[19] = (width >> 8)  & 0x000000ff;
	header[20] = (width >> 16) & 0x000000ff;
	header[21] = (width >> 24) & 0x000000ff;

	// height
	header[22] = height &0x000000ff;
	header[23] = (height >> 8)  & 0x000000ff;
	header[24] = (height >> 16) & 0x000000ff;
	header[25] = (height >> 24) & 0x000000ff;

	// bit per pixel
	header[28] = bit_per_pixel;

	// write header
	fwrite(header, sizeof(unsigned char), rgb_raw_data_offset, fp_t);

	// write image
	fwrite(image_t, sizeof(unsigned char), (size_t)(long)width * height * byte_per_pixel, fp_t);

	fclose(fp_s);
	fclose(fp_t);

	return 0;
}

int
init_device ()
{	// Task 1: Device (GPU) Initialization //Done!
	// Hint  : cudaSetDevice()
	hipSetDevice(0);
	return 0;
}

int
main(int argc, char **argv) {
	init_device();

	const char *input = "candy.bmp";
	if (argc > 1) input = argv[1];
	read_bmp(input); // 24 bit gray level image
		
	// Task 1: Allocate memory on GPU //Done!!
	// Hint  : cudaMalloc ()
	//         What do we need to store on GPU? (input image, output image, ...)
	unsigned char  *d_image_s = NULL;     // source image array
	unsigned char  *d_image_t = NULL;     // target image array
	int *d_mask = NULL; // mask array
  hipMalloc((void**)&d_image_t, (size_t)width * height * byte_per_pixel);
  hipMalloc((void**)&d_image_s, (size_t)width * height * byte_per_pixel);
	hipMalloc((void**)&d_mask, (size_t)sizeof(int) * MASK_N * MASK_Y * MASK_X);
	// Task 1: Memory copy from Host to Device (GPU) //Done!!
	// Hint  : cudaMemcpy ( ... , cudaMemcpyHostToDevice )
	hipMemcpy(d_image_s, image_s, width * height * byte_per_pixel, hipMemcpyHostToDevice);
	hipMemcpy(d_mask, mask, sizeof(int) * MASK_N * MASK_Y * MASK_X, hipMemcpyHostToDevice);
	// Task 1: Modify sobel() to CUDA kernel function //Done!
	// Hint  : sobel_Kernel <<< ??? , ??? >>> ( ??? );
	sobel<<<height, 256>>>( d_image_s, d_image_t, width, height, byte_per_pixel, d_mask);
	// Task 1: Memory Copy from Device (GPU) to Host //DOne!
	// Hint  : cudaMemcpy ( ... , cudaMemcpyDeviceToHost )
	hipMemcpy(image_t, d_image_t, (size_t)width * height * byte_per_pixel, hipMemcpyDeviceToHost);
	// Task 1: Free memory on device //Done!
	// Hint  : cudaFree ( ... )
	hipFree(d_image_t);
  hipFree(d_image_s);
  hipFree(d_mask);

	write_bmp("out.bmp");

	// Task 3: Free Pinned memory //Done!
	// Hint  : replace free ( ... ) by cudaFreeHost ( ... )
	hipHostFree (image_s);
	hipHostFree (image_t);

}
